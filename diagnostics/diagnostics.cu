#include "hip/hip_runtime.h"
#include <math.h>
#include <iostream>
#include <hip/hip_cooperative_groups.h>

#include "diagnostics.h"

using namespace std;
namespace cg = cooperative_groups;

// This kernel computes the length (L) of a single cell
// it does so and writes it to its block-specific shared
// memory view before reducing it on global memory
__global__ void singleCellInterfaceLength(double *phi, double *block_results, const int nx, const int ny, const double dx, const double dy, const int unidimensional_size, const double epsilon) {
    // handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    // dynamically allocated shared memory
    // its size is given when running the
    // kernel
    extern __shared__ double sdata[];

    // compute unique thread index
	int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // don't handle non-existent indexes
    if (i >= unidimensional_size) {
        sdata[tid] = 0;
    }

    // compute interface length
    double phi_x = (phi[i + 1] - phi[i - 1]) / 2.0 / dx; 
    double phi_y = (phi[i + nx] - phi[i - nx]) / 2.0 / dy; 
    // compute the norm of gradient: norm(grad(phi)) 
    double normGrad = sqrt(phi_x * phi_x + phi_y * phi_y);
    // compute the dirac function approximation
    double delta = (1.0 / sqrt(2.0 * M_PI * epsilon)) * exp( - (phi[i] * phi[i]) / (2.0 * epsilon));
    // L = delta * norm(grad(phi)) * dx * dy
    // put data in shared memory
    sdata[tid] = delta * normGrad * dx * dy;

    // synchronize all threads in block
    // to ensure they have all computed
    // and stored their length in shared
    // memory
    cg::sync(cta);

    // do reduction in shared memory
    // it is done in log_2(n) operations
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        cg::sync(cta);
    }

    // write the block result to global memory
    if (tid == 0) block_results[blockIdx.x] = sdata[0];
}

// The total interface length (L) is computed by following the following algorithm
// L ~ sum_{i,j} delta(phi_{i,j}) norm (grad(phi)) dx dy
// with delta(phi) an approximation of the dirac function := delta(phi) = 1 / sqrt (2 * pi * epsilon) * exp (- phi*phi / 2 / epsilon) 
void computeInterfaceLength(double** phi, const int nx, const int ny, const double dx, const double dy){
    // Fixed parameter for the dirac function
    double epsilon = 0.001;
    
    // reduce phi to one dimension
    const int unidimensional_size = nx * ny;
    double* phi_n = new double[unidimensional_size];
    
    for (int i = 0; i < unidimensional_size; i++) {
        // compute two dimensional index
        int ii = i % nx;
        int jj = floor(i / nx);

        // assign value to copy of phi
        phi_n[i] = phi[ii][jj];
    }

    // allocate memory on the device
    // for host-scoped data
    const int N_THREADS = 1024;
    const int N_BLOCKS = ceil((double)(unidimensional_size)/N_THREADS);

    size_t unidimensional_size_bytes = unidimensional_size * sizeof(double);
    double *d_phi_n, *h_block_results, *d_block_results;
    // create host-scoped
    // individual blocks result
    h_block_results = new double[N_BLOCKS];

    hipMalloc((void **)&d_phi_n, unidimensional_size_bytes);
    hipMalloc((void **)&d_block_results, N_BLOCKS * sizeof(double));

    // copy data to device memory
    hipMemcpy(d_phi_n, phi_n, unidimensional_size_bytes, hipMemcpyHostToDevice);

    // launch kernel with shared memory size
    size_t shared_memory_size = N_THREADS * sizeof(double);
    singleCellInterfaceLength<<<N_BLOCKS, N_THREADS, shared_memory_size>>>(d_phi_n, d_block_results, nx, ny, dx, dy, unidimensional_size, epsilon);
    hipDeviceSynchronize();

    // copy block results from
    // device back to host
    hipMemcpy(h_block_results, d_block_results, N_BLOCKS * sizeof(double), hipMemcpyDeviceToHost);

    // final reduction on the host
    double length = 0.0;
    for (int i = 0; i < N_BLOCKS; i++) {
        length += h_block_results[i];
    }

    // free memory on device
	hipFree(d_phi_n);
	hipFree(d_block_results);

    // deallocate memory
    delete[] phi_n;
    delete[] h_block_results;

    // Print the total interface length 
    cout << "The total interface length is " << length << " m\n";
}


// The interface curvature (L) is computed by following the following algorithm
// curvature = (phi_xx * phi_y **2 - 2.0 * phi_x * phi_y * phi_xy + phi_yy * phi_x **2) / (phi_x **2 + phi_y **2) ** (3/2) 
// with phi_x:= d phi / dx, phi_y:= d phi / dy 
// and phi_xx:= d phi_x / dx, phi_yy:= d phi_y / dy, phi_xy:= d phi_x / dy 
void computeInterfaceCurvature(double** phi,double** curvature, const int nx, const int ny, const double dx, const double dy){

    double maxCurvature = 0.0;

    for (int i = 1; i < nx-1; i++){
        for (int j = 1; j < ny-1; j++){

            if (abs(phi[i][j]) < 3.0 * dx ){ //Compute the curvature only near the interface 

                // first derivative
                double phi_x = (phi[i+1][j]-phi[i-1][j])/2.0/dx;
                double phi_y = (phi[i][j+1]-phi[i][j-1])/2.0/dy;

                // second derivative
                double phi_xx = (phi[i+1][j]-2.0*phi[i][j]+phi[i-1][j])/dx/dx;
                double phi_yy = (phi[i][j+1]-2.0*phi[i][j]+phi[i][j-1])/dy/dy;
                double phi_xy = (phi[i+1][j+1] - phi[i+1][j-1] - phi[i-1][j+1] + phi[i-1][j-1])/dx/dy/4.0;

                // compute curvature
                curvature[i][j] = (phi_xx*phi_y*phi_y - 2.0* phi_x*phi_y*phi_xy + phi_yy*phi_x*phi_x)/
                    pow(phi_x*phi_x+phi_y*phi_y,1.5);

                // Replace the maximum curvature
                if (abs(curvature[i][j]) > maxCurvature){maxCurvature = abs(curvature[i][j]);}

            }
            else {// Default value if the cell is not closed to the interface 
                curvature[i][j] = 0.0;
            }
        }
    }

    // Print the maximum interface curvature 
    cout << "The maximum curvature is " << maxCurvature << " m^{-2}\n";
}