#include "hip/hip_runtime.h"
#include "solve.h"

#include <hip/hip_runtime.h>
#include <cmath>

using namespace std;

// This kernel computes φ(t+Δt) of a single cell
// based on its horizontal speed u, vertical speed
// v and its cardinal neighbors
__global__ void singleCellEquationExplicit(double *phi, double *phi_n, double *u, double *v, const double dt, const double dx, const double dy, const int nxy, const int size) {
    // compute unique thread index
	int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // don't handle non-existent indexes
    // due to 2d ceiling necessity
    if (i > size) {
        return;
    }

    // compute two-dimensional index
    // for horizontal and vertical
    // boundary checking -> don't handle
    // if it is on boundary
    int ii = i % nxy;
    if (ii == 0 || ii == (nxy - 1)) {
        return;
    }
    int jj = floor((double)i / nxy);
    if (jj == 0 || jj == (nxy - 1)) {
        return;
    }

    // compute φ(t+Δt)
    phi[i] = phi_n[i];
    
    if (u[i] < 0.0) {
        phi[i] -= dt * (u[i]*(phi_n[i+1] - phi_n[i])/dx);
    }
    else {
        phi[i] -= dt * (u[i]*(phi_n[i] - phi_n[i-1])/dx);
    }

    if (v[i] < 0.0) {
        phi[i] -= dt * (v[i]*(phi_n[i+nxy] - phi_n[i])/dy);
    }
    else {
        phi[i] -= dt * (v[i]*(phi_n[i] - phi_n[i-nxy])/dy);
    } 
}

// Compute the boundaries of the domain for the phi field
void computeBoundaries(double* phi, const int nx, const int ny){
    // Upper and Lower boundaries (extrapolation)
    for (int i = 0; i < nx ; i++){
        phi[i] = 2.0 * phi[i + nx] - phi[i + 2 * nx];
        phi[i + nx * (ny - 1)] = 2.0 * phi[i + nx * (ny - 2)] - phi[i + nx * (ny - 3)];
    }

    // Left and Right boundaries (extrapolation)
    for (int j = 0; j < ny; j++){
        phi[j * nx] = 2.0 * phi[1 + j * nx] - phi[2 + j * nx];
        phi[(j * nx) + nx - 1] = 2.0 * phi[(j * nx) + nx - 2] - phi[(j * nx) + nx - 3];
    }
}


// Solving advection equation on the inside domain ([1;nx-2] x [1;ny-2])
// Equation solved: d phi / dt + u d phi / dx + v d phi / dy = 0
// Using the euler explicit numerical scheme => phi = phi_n - (u d phi / dx + v d phi / dy)
// A first order upwind scheme is used to stabilize the solver (https://en.wikipedia.org/wiki/Upwind_scheme)
void solveAdvectionEquationExplicit(
    double* phi, double* u, double* v, const int nx, const int ny, const double dx, const double dy, const double dt){

    const int unidimensional_size = nx * ny;
    double* phi_n = new double[unidimensional_size];
    
    for (int i = 0; i < unidimensional_size; i++) {
        // assign value to copy of phi
        phi_n[i] = phi[i];
    }

    // Compute the advection equation 
    
    // allocate memory on the device
    // for host-scoped data
    size_t unidimensional_size_bytes = unidimensional_size * sizeof(double);


    hipMemcpy(d_phi_n, phi_n, unidimensional_size_bytes, hipMemcpyHostToDevice);

    const int N_THREADS = 1024;
    const int N_BLOCKS = ceil((double)(unidimensional_size)/N_THREADS);

	singleCellEquationExplicit<<<N_BLOCKS, N_THREADS>>>(d_phi, d_phi_n, d_u, d_v, dt, dx, dy, nx, unidimensional_size);
    hipDeviceSynchronize();

    // copy result from device back to host
	hipMemcpy(phi_n, d_phi, unidimensional_size_bytes, hipMemcpyDeviceToHost);


    // copy back to array
    for (int i = 0; i < unidimensional_size; i++) {
        phi[i] = phi_n[i];
    }

    // Refresh the boundaries values
    computeBoundaries(phi, nx, ny);

    // Deallocate memory
    delete[] phi_n;
}